#include "hip/hip_runtime.h"
// Copyright (c) 2020 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#pragma once
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/operators/index_select_op.h"
#include "paddle/fluid/platform/cuda_primitives.h"

namespace paddle {
namespace operators {

using platform::PADDLE_CUDA_NUM_THREADS;
using Tensor = framework::Tensor;
using LoDTensor = framework::LoDTensor;

template <typename T>
__global__ void index_select_cuda_kernel(const T* input, T* output,
                                         int64_t* index, int64_t N,
                                         int64_t stride, int64_t size) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t dim_idx = idx % (stride * size) / stride;
  int64_t src_dim_idx = index[dim_idx];
  int64_t input_idx = idx + (src_dim_idx - dim_idx) * stride output[idx] =
                          input[input_idx];
}

template <typename T>
__global__ void index_select_cuda_kernel_int(const T* input, T* output,
                                             int* index, int64_t N,
                                             int64_t stride, int64_t size) {
  int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) {
    return;
  }

  int64_t dim_idx = idx % (stride * size) / stride;
  int src_dim_idx = index[dim_idx];
  int64_t input_idx = idx + (src_dim_idx - dim_idx) * stride output[idx] =
                          input[input_idx];
}

template <typename DeviceContext, typename T>
class IndexSelectCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto* in = context.Input<LoDTensor>("X");
    auto* index = context.Input<LoDTensor>("Index");
    auto* out = context.Output<LoDTensor>("Out");
    int dim = context.Attr<int>("dim");
    auto input_dim = in->dims();
    dim = dim >= 0 ? dim : dim + input_dim.size();
    auto stride_dim = framework::stride(input_dim) int64_t stride =
        stride_dim[dim];
    int64_t size = input_dim[dim];

    const auto& index_type = index->type();
    bool index_type_match = index_type == framework::proto::VarType::INT64 ||
                            index_type == framework::proto::VarType::INT32;
    PADDLE_ENFORCE_EQ(index_type_match, true,
                      platform::errors::InvalidArgument(
                          "Input(Index) holds the wrong type, it holds %s, but "
                          "desires to be %s or %s",
                          paddle::framework::DataTypeToString(index_type),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT32),
                          paddle::framework::DataTypeToString(
                              framework::proto::VarType::INT64)));

    auto* in_data = in->data<T>();
    auto* out_data = out->mutable_data<T>(context.GetPlace());
    int64_t numel = out_data->numel();

    auto stream =
        context.template device_context<platform::CUDADeviceContext>().stream();

    if (index_type == framework::proto::VarType::INT64) {
      int64_t* index_data = index->data<int64_t>();
      index_select_cuda_kernel<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                                     PADDLE_CUDA_NUM_THREADS,
                                 PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          in_data, out_data, index_data, numel, stride, size);
    } else {
      int* index_data = index->data<int>();
      index_select_cuda_kernel_int<<<(numel + PADDLE_CUDA_NUM_THREADS - 1) /
                                         PADDLE_CUDA_NUM_THREADS,
                                     PADDLE_CUDA_NUM_THREADS, 0, stream>>>(
          in_data, out_data, index_data, numel, stride, size);
    }
  }
};

template <typename DeviceContext, typename T>
class IndexSelectGradCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {}
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(
    index_select,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::IndexSelectCUDAKernel<paddle::platform::CUDADeviceContext, int64_t>);
REGISTER_OP_CUDA_KERNEL(
    index_select_grad,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, float>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, double>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext, int>,
    ops::IndexSelectGradCUDAKernel<paddle::platform::CUDADeviceContext,
                                   int64_t>);
